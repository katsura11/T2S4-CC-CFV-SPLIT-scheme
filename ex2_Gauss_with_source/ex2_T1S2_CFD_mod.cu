#include <iterator>
#include "hip/hip_runtime.h"

#include <cstdio>
#include <ctime>

#include <iostream>
#include <math.h>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iomanip>
#include <fstream>
#include <device_atomic_functions.h>

#define x0 -0.35
#define y0  0.0
#define sigma 0.005
#define X_min -1.0
#define X_max  1.0
#define Y_min -1.0
#define Y_max  1.0

using namespace std;

//************************************
// Definition of variable types
//************************************
typedef int INT;
typedef double FLOAT;

//check error
#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)

//************** Inline function **********************
inline FLOAT Vx(FLOAT x, FLOAT y) { return -M_PI*cos(0.5*M_PI*x)*sin(0.5*M_PI*y); }
inline FLOAT Vy(FLOAT x, FLOAT y) { return  M_PI*cos(0.5*M_PI*y)*sin(0.5*M_PI*x); }
inline FLOAT C0(FLOAT x, FLOAT y) {
  return exp(-(pow((x - x0), 2) + pow((y - y0), 2)) / sigma);
}
inline FLOAT C_Exact(FLOAT x, FLOAT y, FLOAT t, FLOAT K) {
    FLOAT x_star =  x * cos(M_PI * t) + y * sin(M_PI * t);
    FLOAT y_star = -x * sin(M_PI * t) + y * cos(M_PI * t);
    return sigma / (sigma + 4 * K * t) *
            exp(-(pow((x_star - x0), 2) + pow((y_star - y0), 2)) / (sigma + 4 * K * t));
}

inline FLOAT sum(FLOAT *v_f, INT N_grid) {
    FLOAT sum = 0.0;
    for (INT i = 0; i < N_grid; i++) sum += v_f[i];
    return sum;
}

//****************************** Device function **********************
__device__ FLOAT d_Vx(FLOAT x, FLOAT y) { return -M_PI*cos(0.5*M_PI*x)*sin(0.5*M_PI*y); }
__device__ FLOAT d_Vy(FLOAT x, FLOAT y) { return  M_PI*cos(0.5*M_PI*y)*sin(0.5*M_PI*x); }
__device__ FLOAT d_C0(FLOAT x, FLOAT y) {
    return exp(-(pow((x - x0), 2) + pow((y - y0), 2)) / sigma);
}
__device__ FLOAT d_C_Exact(FLOAT x, FLOAT y, FLOAT t, FLOAT K) {
    FLOAT x_star =  x * cos(M_PI * t) + y * sin(M_PI * t);
    FLOAT y_star = -x * sin(M_PI * t) + y * cos(M_PI * t);
    return sigma / (sigma + 4 * K * t) *
           exp(-(pow((x_star - x0), 2) + pow((y_star - y0), 2)) / (sigma + 4 * K * t));
}
__device__ FLOAT d_f(FLOAT x, FLOAT y, FLOAT t, FLOAT K) {
    FLOAT x_star =  x * cos(M_PI * t) + y * sin(M_PI * t);
    FLOAT y_star = -x * sin(M_PI * t) + y * cos(M_PI * t);
    FLOAT p1 = sigma / (sigma + 4 * K * t) *
               exp(-(pow((x_star - x0), 2) + pow((y_star - y0), 2)) / (sigma + 4 * K * t));
    FLOAT p2 = (x - cos(0.5*M_PI*y) * sin(0.5*M_PI*x)) * ( x0*sin(M_PI*t) + y0*cos(M_PI*t) - y) 
             + (y - cos(0.5*M_PI*x) * sin(0.5*M_PI*y)) * (-x0*cos(M_PI*t) + y0*sin(M_PI*t) + x);
    return - M_PI * p1 * p2 /(sigma + 4 * K * t); 
}

// Output the result to a file
void output_result(FLOAT *vec, INT t, INT Nx, INT Ny, FLOAT dt) {
    FILE *fp;
    char sfile[256];
    int i_cell, j_cell;
    FLOAT x, y;
    FLOAT dx, dy;

    dx = (X_max - X_min) / Nx;
    dy = (Y_max - Y_min) / Ny;

    sprintf(sfile, "data_T1S2_%06d.txt", t);
    fp = fopen(sfile, "w");
    //fprintf(fp, "#time = %lf\n", (double)t * dt);
    //fprintf(fp, "#x y u\n");
    for (i_cell = 0; i_cell < Nx; i_cell++) {
        x = X_min + dx * (i_cell + 0.5);
        for (j_cell = 0; j_cell < Ny; j_cell++) {
        y = Y_min + dy * (j_cell + 0.5);
        //fprintf(fp, "%.6lf %.6lf %.4e\n", x, y, vec[i_cell * Ny + j_cell]);
        fprintf(fp, "%.15e\n", vec[i_cell * Ny + j_cell]);
        }
        fprintf(fp, "\n");
    }
    fclose(fp);
    return;
}

//****************************** kernel function **********************
__global__ void x_create_matrix_on_gpu(FLOAT *v_d_coA, FLOAT *v_d_coB, FLOAT *v_d_coC, FLOAT *v_d_b, 
									    FLOAT *v_d_C_xphf, FLOAT *v_d_C0, FLOAT *v_d_C0_bar, 
                                        FLOAT *v_d_f0, FLOAT dx, FLOAT dy, FLOAT t, FLOAT dt, INT Nx, 
                                        INT Ny, INT N_grid, FLOAT Kx, FLOAT dt_mul_dx_square_inverse)
{
	int ind = blockDim.x * blockIdx.x + threadIdx.x;
	FLOAT x, y, x_bar, x_m, ksi;
	int i, j, m;
	while (ind < N_grid)
	{
		i = ind / Ny ;
		j = ind % Ny ;
		x = X_min + dx * (i + 0.5);
		y = Y_min + dy * (j + 0.5);
		x_bar = x - d_Vx(x, y) * dt;
		m = floor((x_bar - X_min) / dx);
		x_m = X_min + dx * (m + 0.5);
		ksi = (x_bar - x_m)/dx;
		
		//*********************** calculate U_bar **************************
		if (m > 0 && m < Nx - 1)
		{
			v_d_C0_bar[ind] = 0.5 * v_d_C0[(m - 1) * Ny + j] * (ksi - 1) * ksi 
                            - v_d_C0[m * Ny + j] * (ksi - 1) * (ksi + 1) 
			                + 0.5 * v_d_C0[(m + 1) * Ny + j] * (ksi + 1) * ksi ;
		}
        else {
            v_d_C0_bar[ind] = d_C_Exact(x, y, t - dt, Kx);
        }
		v_d_f0[ind] =  d_f(x, y, t, Kx);

        //********************* assemble matrix elements ******************************
		v_d_coA[ind] = -Kx * dt_mul_dx_square_inverse;			       // lower-diagonal
        v_d_coB[ind] = 1.0 + 2.0 * Kx * dt_mul_dx_square_inverse;     // main-diagonal
        v_d_coC[ind] = -Kx * dt_mul_dx_square_inverse;			     // upper-diagonal
        v_d_b[ind] = v_d_C0_bar[ind] + dt * v_d_f0[ind];

		ind += gridDim.x * blockDim.x;
	}
	return;
}

__global__ void y_create_matrix_on_gpu(FLOAT *v_d_coA, FLOAT *v_d_coB, FLOAT *v_d_coC, FLOAT *v_d_b, 
                                      FLOAT *v_d_Cp1, FLOAT *v_d_C_xphf, FLOAT *v_d_C_xphf_bar, 
                                      FLOAT *v_d_f1, FLOAT dx, FLOAT dy, FLOAT t, FLOAT dt, INT Nx, 
                                      INT Ny, INT N_grid, FLOAT Ky, FLOAT dt_mul_dy_square_inverse)
{
	int ind = blockDim.x * blockIdx.x + threadIdx.x;
	FLOAT x, y, y_bar, y_m, ksi;
	int i, j, m;
	while (ind < N_grid)
	{
		i = ind / Ny ;
		j = ind % Ny ;
		x = X_min + dx * (i + 0.5);
		y = Y_min + dy * (j + 0.5);
		y_bar = y - d_Vy(x, y) * dt;
		m = floor((y_bar - Y_min) / dy);
		y_m = Y_min + dy * (m + 0.5);
		ksi = (y_bar - y_m)/dy;

		//************************* calculate U_bar ************************
		if (m > 0 && m < Ny - 1)
		{
			v_d_C_xphf_bar[ind] = 0.5 * v_d_C_xphf[i * Ny + m - 1] * (ksi - 1) * ksi 
                                - v_d_C_xphf[i * Ny + m] * (ksi - 1) * (ksi + 1)
			                 	+ 0.5 * v_d_C_xphf[i * Ny + m + 1] * (ksi + 1) * ksi;
		}
        else {
            v_d_C_xphf_bar[ind] = d_C_Exact(x, y, t - dt, Ky);
        }
        v_d_f1[ind] =  d_f(x, y, t, Ky);

        //******************* assemble matrix elements *************************
		v_d_coA[ind] = -Ky * dt_mul_dy_square_inverse;		          // lower-diagonal
        v_d_coB[ind] = 1.0 + 2.0 * Ky * dt_mul_dy_square_inverse;    // main-diagonal
        v_d_coC[ind] = -Ky * dt_mul_dy_square_inverse;		        // upper-diagonal														 
        v_d_b[ind] = v_d_C_xphf_bar[ind] + dt * v_d_f1[ind];

		ind += gridDim.x * blockDim.x;
	}
	return;
}

__global__ void x_Thomas_on_gpu(FLOAT *v_d_coA, FLOAT *v_d_coB, FLOAT *v_d_coC,
                                FLOAT *v_d_b, FLOAT *v_d_C_xphf, FLOAT *p,
                                FLOAT *q, INT N_grid, INT Nx, INT Ny) {

  INT ins = blockDim.x * blockIdx.x + threadIdx.x;
  INT i_cell, j_cell;
  FLOAT denom;
  while (ins < Ny) {
    j_cell = ins;

    p[j_cell] = v_d_coC[j_cell] / v_d_coB[j_cell];
    q[j_cell] = (v_d_b[j_cell]) / v_d_coB[j_cell];

    for (i_cell = 1; i_cell < Nx; i_cell++) {
        denom = 1.0f / (v_d_coB[i_cell * Ny + j_cell] 
                    - p[(i_cell - 1) * Ny + j_cell] * v_d_coA[i_cell * Ny + j_cell]);
        p[i_cell * Ny + j_cell] = v_d_coC[i_cell * Ny + j_cell] * denom;
        q[i_cell * Ny + j_cell] = (v_d_b[i_cell * Ny + j_cell] 
                                    - q[(i_cell - 1) * Ny + j_cell] * v_d_coA[i_cell * Ny + j_cell]) * denom;
    }

    v_d_C_xphf[(Nx - 1) * Ny + j_cell] = q[(Nx - 1) * Ny + j_cell];
    for (int i_cell = Nx-2; i_cell >= 0; i_cell--) {
        v_d_C_xphf[i_cell * Ny + j_cell] = q[i_cell * Ny + j_cell] 
                                        - p[i_cell * Ny + j_cell] * v_d_C_xphf[(i_cell + 1) * Ny + j_cell];
    }    
    ins += gridDim.x * blockDim.x;
  }
  return;
}

__global__ void y_Thomas_on_gpu(FLOAT *v_d_coA, FLOAT *v_d_coB, FLOAT *v_d_coC,
                                FLOAT *v_d_b, FLOAT *v_d_C_yp1, FLOAT *p,
                                FLOAT *q, INT N_grid, INT Nx, INT Ny) {

  int ins = blockDim.x * blockIdx.x + threadIdx.x;
  int i_cell, j_cell;
  FLOAT denom;

  while (ins < Nx) {
    i_cell = ins ;

    // q[0]=q[0]/b[0] ; d[0]=d[0]/b[0];
    p[i_cell * Ny] = v_d_coC[i_cell * Ny] / v_d_coB[i_cell * Ny];
    q[i_cell * Ny] = (v_d_b[i_cell * Ny]) / v_d_coB[i_cell * Ny];

    for (int j_cell = 1; j_cell < Ny; j_cell++) {
        denom = 1.0f / (v_d_coB[i_cell * Ny + j_cell] 
                    - p[i_cell * Ny + j_cell - 1] * v_d_coA[i_cell * Ny + j_cell]);
        p[i_cell * Ny + j_cell] = v_d_coC[i_cell * Ny + j_cell] * denom;
        q[i_cell * Ny + j_cell] = (v_d_b[i_cell * Ny + j_cell] 
                                - q[i_cell * Ny + j_cell - 1] * v_d_coA[i_cell * Ny + j_cell]) * denom;
    }

    v_d_C_yp1[i_cell * Ny + Ny-1] = q[i_cell * Ny + Ny-1];
    for (int j_cell = Ny-2; j_cell >= 0; j_cell--) {
        v_d_C_yp1[i_cell * Ny + j_cell] = q[i_cell * Ny + j_cell] 
                                        - p[i_cell * Ny + j_cell] * v_d_C_yp1[i_cell * Ny + (j_cell + 1)];
    }  
    ins += gridDim.x * blockDim.x;
  }
  return;
}

//*********************************
// Main Code
//*********************************
int main(int argc, char *argv[]) {
    // Start of the time controller
    clock_t start = clock();

    //**************** initialize variables ******************
    FLOAT T_min, T_max; 
    FLOAT Kx, Ky;
    FLOAT dx, dy, dt;
    FLOAT x, y, t0, t1;
    INT i_cell, j_cell, itime, tid; // 循环变量
    INT Nx, Ny, Nt, T_span, N_grid, N_points;
    FLOAT E_2, E_Inf, Error, Cmassdif, cmass_initial, cmass_end;
    FLOAT *v_h_f, *v_h_C0, *v_h_C1, *v_h_C_Exact;
    FLOAT *v_d_C0, *v_d_C0_bar, *v_d_C_xphf, *v_d_C_xphf_bar, *v_d_Cp1; 
    FLOAT *v_d_coA, *v_d_coB, *v_d_coC, *v_d_b, *p, *q;
    FLOAT *v_d_f0, *v_d_f1;


    if (argc < 2) {
        cout << "please input config file name" << endl;
    }
    //*************************************************
    // read  parameters from config file
    //*************************************************
    ifstream configFile;
    configFile.open(argv[1]);
    string strLine;
    string strKey, strValue;
    size_t pos;
    if (configFile.is_open()) {
        cout << "open config file ok" << endl;
        while (!configFile.eof()) {
        getline(configFile, strLine);
        pos = strLine.find(':');
        strKey = strLine.substr(0, pos);
        strValue = strLine.substr(pos + 1);
        if (strKey.compare("T_min") == 0) {
            sscanf(strValue.c_str(), "%lf", &T_min);
        }
        if (strKey.compare("T_max") == 0) {
            sscanf(strValue.c_str(), "%lf", &T_max);
        }
        if (strKey.compare("N") == 0) {
            sscanf(strValue.c_str(), "%d", &Nx);
        }
        if (strKey.compare("N") == 0) {
            sscanf(strValue.c_str(), "%d", &Ny);
        }
        if (strKey.compare("Nt") == 0) {
            sscanf(strValue.c_str(), "%d", &Nt);
        }
        if (strKey.compare("T_span") == 0) {
            sscanf(strValue.c_str(), "%d", &T_span);
        }
        if (strKey.compare("K") == 0) {
            sscanf(strValue.c_str(), "%lf", &Kx);
        }
        if (strKey.compare("K") == 0) {
            sscanf(strValue.c_str(), "%lf", &Ky);
        }
        }
    } else {
        cout << "Cannot open config file!" << endl;
        return 1;
    }
    configFile.close();
    //fclose(setup_file);

    // Calculate delta_x, delta_y, delta_t, and the problem size 
    dx = (X_max - X_min) / Nx;
    dy = (Y_max - Y_min) / Ny;
    dt = (T_max - T_min) / Nt;

    N_grid = Nx * Ny;
    N_points = (Nx + 1) * (Ny + 1);

    FLOAT dt_half = 0.5 * dt;
    FLOAT dt_mul_dx_square_inverse = dt / (dx * dx);
    FLOAT dt_mul_dy_square_inverse = dt / (dy * dy);
    

    // GPU related variables and parameters
    int numBlocks;       // Number of blocks
    int threadsPerBlock; // Number of threads
    int maxThreadsPerBlock;
    hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        //maxThreadsPerBlock = prop.maxThreadsPerBlock;
        maxThreadsPerBlock = 256;
        if (N_points < maxThreadsPerBlock)
        {
            threadsPerBlock = N_points;
            numBlocks = 1;
        }
        else
        {
            threadsPerBlock = maxThreadsPerBlock;
            numBlocks = (N_points + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
        }

    // Allocate host memory for the location vector
    v_h_f = (FLOAT *)malloc(N_grid * sizeof(FLOAT));
    v_h_C0 = (FLOAT *)malloc(N_grid * sizeof(FLOAT));
    v_h_C1 = (FLOAT *)malloc(N_grid * sizeof(FLOAT));
    v_h_C_Exact = (FLOAT *)malloc(N_grid * sizeof(FLOAT));

    // Create device vector to store the solution 
    hipMalloc((void **)&v_d_C0, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_Cp1, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_C0_bar, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_C_xphf, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_C_xphf_bar, N_grid * sizeof(FLOAT));

    // Create device vector to store matrix elements
    hipMalloc((void **)&p, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&q, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_b, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_coA, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_coB, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_coC, N_grid * sizeof(FLOAT));

    // Create device vector to store source term
    hipMalloc((void **)&v_d_f0, N_grid * sizeof(FLOAT));
    hipMalloc((void **)&v_d_f1, N_grid * sizeof(FLOAT));


    // Calculte Initial condition and Exact solution on CPU
    clock_t t2 = clock();
    for (i_cell = 0; i_cell < Nx; i_cell++){
        x = X_min + (i_cell + 0.5) * dx;
        for (j_cell = 0; j_cell < Ny; j_cell++){
            y = Y_min + (j_cell + 0.5) * dy;
            tid = i_cell * Ny + j_cell;       
            v_h_C0[tid] = C_Exact(x, y, 0, Kx);
            v_h_C_Exact[tid] = C_Exact(x, y, T_max, Kx);
        }
    } 
    hipMemcpy(v_d_C0, v_h_C0, N_grid * sizeof(FLOAT), hipMemcpyHostToDevice);
    output_result(v_h_C0, 0, Nx, Ny, dt);


    FLOAT sumf = 0.0;
    for (itime = 1; itime <= Nt; itime++){
        //cout << "itime = " << itime << endl;
        t1 = itime * dt;
        //**************************** step 1**********************************
        x_create_matrix_on_gpu<<<numBlocks, threadsPerBlock>>>
        (v_d_coA, v_d_coB, v_d_coC, v_d_b, v_d_C_xphf, v_d_C0, v_d_C0_bar, v_d_f0, 
        dx, dy, t1, dt, Nx, Ny, N_grid, Kx, dt_mul_dx_square_inverse);
        hipDeviceSynchronize();

        hipMemcpy(v_h_f, v_d_f0, N_grid * sizeof(FLOAT), hipMemcpyDeviceToHost);
        sumf += sum(v_h_f, N_grid)* dx * dy * dt;

        x_Thomas_on_gpu<<<numBlocks, threadsPerBlock>>>
        (v_d_coA, v_d_coB, v_d_coC, v_d_b, v_d_C_xphf, p, q, N_grid, Nx, Ny);
        hipDeviceSynchronize();

        //***************************** step 2 *********************************
        y_create_matrix_on_gpu<<<numBlocks, threadsPerBlock>>>
        (v_d_coA, v_d_coB, v_d_coC, v_d_b, v_d_Cp1, v_d_C_xphf, v_d_C_xphf_bar, v_d_f1, 
        dx, dy, t1, dt, Nx, Ny, N_grid, Ky, dt_mul_dx_square_inverse);
        hipDeviceSynchronize();

        hipMemcpy(v_h_f, v_d_f1, N_grid * sizeof(FLOAT), hipMemcpyDeviceToHost);
        sumf += sum(v_h_f, N_grid)* dx * dy * dt;

        y_Thomas_on_gpu<<<numBlocks, threadsPerBlock>>>
        (v_d_coA, v_d_coB, v_d_coC, v_d_b, v_d_Cp1, p, q, N_grid, Nx, Ny);
        hipDeviceSynchronize();

        //*************************** update solution ***************************
        hipMemcpy(v_d_C0, v_d_Cp1, N_grid * sizeof(FLOAT), hipMemcpyDeviceToDevice);
        
        // 输出数据
        if (itime % T_span == 0) {
            hipMemcpy(v_h_C1, v_d_C0, N_grid * sizeof(FLOAT), hipMemcpyDeviceToHost);
            output_result(v_h_C1, itime, Nx, Ny, dt);
            /*
            Cmassdif = 0.0;
            cmass_end = 0.0;
            cmass_initial = 0.0;
            for (tid = 0; tid < N_grid; tid++) {
                cmass_end = cmass_end + v_h_C1[tid] * dx * dy;
                cmass_initial = cmass_initial + v_h_C0[tid] * dx * dy;
            }
            Cmassdif = cmass_initial - cmass_end - sumf * dx * dy;
            printf("time: %4lf, Cmassdif:%.15e, Cmass_end:%.15e\n", itime*dt, Cmassdif, cmass_end);
            */
        }  
    
    }
    hipMemcpy(v_h_C1, v_d_C0, N_grid * sizeof(FLOAT), hipMemcpyDeviceToHost);

    // calculate error
    E_2 = 0.0;
    E_Inf = 0.0;
    Error = 0.0;
    Cmassdif = 0.0;
    cmass_end = 0.0;
    cmass_initial = 0.0;
    
    for (tid = 0; tid < N_grid; tid++) {
        Error = v_h_C1[tid] - v_h_C_Exact[tid];
        E_2 += Error * Error;
        E_Inf = fmax(E_Inf, fabs(Error));
        cmass_end = cmass_end + v_h_C1[tid] * dx * dy;
        cmass_initial = cmass_initial + v_h_C0[tid] * dx * dy;
    }
    Cmassdif = cmass_initial - cmass_end - sumf;
    E_2 = sqrt(E_2 * dx *dy);
    //E_2 = sqrt(E_2) * dx;
    
    cout << "K:" << Kx << endl;
    cout << "dx:" << dx << endl;
    cout << "dy:" << dy << endl;
    cout << "dt:" << dt << endl;
    cout << "Nx:" << Nx << endl;
    cout << "Ny:" << Ny << endl;
    cout << "Nt:" << Nt << endl;
    printf("E2:%.4e\n", E_2); 
    printf("\n");
    printf("EInf:%.4e\n", E_Inf);
    printf("\n");
    printf("Cmassdif:%.15e\n", Cmassdif);
    /*
    printf("sumf:%.15e\n", sumf * dx * dy);
    printf("cmass_end:%.15e\n", cmass_end);
    printf("cmass_initial:%.15e\n", cmass_initial);
    */
      
    clock_t end = clock();
    double time = (end - start) / (double)CLOCKS_PER_SEC;
    cout << "time:" << time << endl;

    hipFree(v_d_C0);
    hipFree(v_d_Cp1);
    hipFree(v_d_b);
    hipFree(v_d_coA);
    hipFree(v_d_coB);
    hipFree(v_d_coC);
    hipFree(v_d_coA);
    hipFree(v_d_coB);
    hipFree(v_d_coC);
    hipFree(v_d_f0);
    hipFree(v_d_f1);
    hipFree(v_d_C0_bar);
    hipFree(v_d_C_xphf);
    hipFree(v_d_C_xphf_bar);
    hipFree(p);
    hipFree(q);

    return 0;
}